#include <iostream>
#include <cmath>
#include <vector>
#include <string>
#include <chrono>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>
#include <sstream>

using namespace std;
using namespace std::chrono;

string input_path = "input/input.txt";

__device__ const double G = 6.674e-11; // гравитационная постоянная
__device__ const double dt = 0.001; // шаг по времени
__device__ const double e = 0.01; // чтобы сила не ушла в бесконечность
__device__ const double t_end = 20; // конечное время
__device__ double t = 0; // начальное время

int block_count = 1; // число блоков (thread_count выбирается динамически)

struct MatPoint { // класс для материальной точки
    double x; // координата по x
    double y; // координата по y
    double vx; // скорость по x
    double vy; // скорость по y
    double m; // масса
};

struct Direction {
    double f_x; // сила по направлению x
    double f_y; // сила по направлению y
};

MatPoint *points; // материальные точки
Direction *forces; // силы по напралениям

__device__ void calcForce(const MatPoint points[], Direction forces[]) { // функция для пересчёта сил, действующих на материальную точку (на вход - вектор точек)
    double sum_x = 0; // сумма сил, действующих на точку в направлении x
    double sum_y = 0; // сумма сил, действующих на точку в направлении y
    for (unsigned i = 0; i < blockDim.x; ++i) { // проход по всем точкам
        if (threadIdx.x == i) { // не рассматриваем одинаковые точки
            continue;
        }
        double dist = sqrt(pow((points[i].x - points[threadIdx.x].x), 2) + pow((points[i].y - points[threadIdx.x].y), 2)); // рассчёт дистанции между точками
        sum_x += points[i].m * (points[i].x - points[threadIdx.x].x) / (pow(dist, 3) + e); // сила по x, действующая на точку, рассматриваемую потоком threadIdx.x со стороны точки i
        sum_y += points[i].m * (points[i].y - points[threadIdx.x].y) / (pow(dist, 3) + e); // сила по y, действующая на точку, рассматриваемую потоком threadIdx.x со стороны точки i
    }
    forces[threadIdx.x].f_x = G * points[threadIdx.x].m * sum_x; // общая сила, действующая на материальную точку threadIdx.x в направлении x
    forces[threadIdx.x].f_y = G * points[threadIdx.x].m * sum_y; // общая сила, действующая на материальную точку threadIdx.x в направлении y
}

// обновление данных точек
__device__ void simulationStep(MatPoint points[], const Direction forces[]) {
    points[threadIdx.x].vx += forces[threadIdx.x].f_x / points[threadIdx.x].m * dt; // изменяем скорость по направлению x
    points[threadIdx.x].vy += forces[threadIdx.x].f_y / points[threadIdx.x].m * dt; // изменяем скорость по направлению y
    points[threadIdx.x].x += points[threadIdx.x].vx * dt; // изменяем положение по x
    points[threadIdx.x].y += points[threadIdx.x].vy * dt; // изменяем положение по y
}

// вывод результатов в формате "время x_1 y_1 x_2 y_2 ..."
//===============================
// __device__ void print_results(double t, const MatPoint points[], string output_path="output/output") {
//     ofstream file(output_path + to_string(threadIdx.x) + ".txt", ios::app);
//     file << t << " " << points[threadIdx.x].x << " " << points[threadIdx.x].y << endl; 
//     file.close();
// }
//-------------------------------
// __device__ void print_results(double t, const MatPoint points[]) {
//     printf("%.3f %f %f\n", t, points[threadIdx.x].x, points[threadIdx.x].y);
// }
//-------------------------------
__device__ void print_results(const double t, const MatPoint points[]) {
    printf("%.3f ", t);
    for(int i=0; i<blockDim.x; ++i)
        printf("%f %f ", points[i].x, points[i].y);
    printf("\n");
}
//===============================


__global__ void Routine(MatPoint points[], Direction forces[]){ // функция, запускаемая на девайсе
    // printf("%f %f %f %f %f\n", points[threadIdx.x].x, points[threadIdx.x].y, points[threadIdx.x].vx, points[threadIdx.x].vy, points[ithreadIdx.x].m);
    while (t < t_end) {
        calcForce(points, forces); // считаем силы, действующие на все точки
        __syncthreads(); // синхронизируемся перед обновлением данных точек
        simulationStep(points, forces); // обновляем данные точек
        __syncthreads(); // синхронизируемся перед следующим шагом
        if (threadIdx.x == 0) { // поток 0 выводит данные и собновляет время
            print_results(t, points); // выводим результат на шаге
            t += dt; // увеличиваем время
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc == 2) // проверка наличия аргумента (сама программа + путь для считывания файла)
        input_path = argv[1];
    // считаем число строк в файле ==> столько будет потоков
    ifstream file1(input_path);
    int thread_count = 0;
    string line;
    while (getline(file1, line))
        ++thread_count;
    file1.close();

    hipMallocManaged(&points, thread_count * sizeof(MatPoint)); // выделение памяти под точки
    hipMallocManaged(&forces, thread_count * sizeof(Direction)); // выделение памяти по действующие силы

    // заполнение массивов точек и действующих сил
    ifstream file2(input_path);
    double x, y, vx, vy, m;
    int iter = 0;
    while (!file2.eof()) { // один поток - одна точка
        file2 >> x >> y >> vx >> vy >> m; // берём данные из файла
        points[iter] = {x, y, vx, vy, m}; // заполняем данные для точек
        forces[iter] = {0, 0}; // заполняем данные для точек
        iter++;
    }
    file2.close();

    auto start = high_resolution_clock::now();

    Routine<<<block_count, thread_count>>>(points, forces); // вызов девайсной функции (передаём число блоков и число потоков в блоке)
    hipDeviceSynchronize();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(stop - start);
    // cout << fixed << setprecision(12) << duration.count() * 1e-9 << endl;

    hipFree(points); // освобождение памяти
    hipFree(forces);

    return 0;
}