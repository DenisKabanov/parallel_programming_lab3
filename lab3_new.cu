#include <iostream>
#include <cmath>
#include <vector>
#include <string>
#include <chrono>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>
#include <sstream>

using namespace std;
using namespace std::chrono;

string input_path = "input/input.txt";

__device__ const double G = 6.674e-11; // гравитационная постоянная
__device__ const double dt = 0.001; // шаг по времени
__device__ const double e = 0.01; // чтобы сила не ушла в бесконечность
__device__ const double t_end = 20; // конечное время
__device__ double t = 0; // начальное время

int block_count = 1; // число блоков (thread_count выбирается динамически)

//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
// в предположении, что число точек делится нацело на число потоков
int points_per_thread = 1; // число точек на поток 
const int max_threads = 32; // лимит на число потоков
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

struct MatPoint { // класс для материальной точки
    double x; // координата по x
    double y; // координата по y
    double vx; // скорость по x
    double vy; // скорость по y
    double m; // масса
};

struct Direction {
    double f_x; // сила по направлению x
    double f_y; // сила по направлению y
};

MatPoint *points; // материальные точки
Direction *forces; // силы по напралениям


//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
__device__ void calcForce(const MatPoint points[], Direction forces[], const int points_start_for_thread, const int points_end_for_thread) { // функция для пересчёта сил, действующих на материальную точку (на вход - вектор точек)
    for (int i = points_start_for_thread; i < points_end_for_thread; ++i) { // цикл по точкам, соответствующим потоку
        double sum_x = 0; // сумма сил, действующих на точку в направлении x
        double sum_y = 0; // сумма сил, действующих на точку в направлении y
        for (unsigned j = 0; j < blockDim.x; ++j) { // проход по всем точкам (blockDim.х - число всех потоков)
            if (i == j) { // не рассматриваем одинаковые точки
                continue;
            }
            double dist = sqrt(pow((points[j].x - points[i].x), 2) + pow((points[j].y - points[i].y), 2)); // рассчёт дистанции между точками
            sum_x += points[j].m * (points[j].x - points[i].x) / (pow(dist, 3) + e); // сила по x, действующая на точку i со стороны точки j
            sum_y += points[j].m * (points[j].y - points[i].y) / (pow(dist, 3) + e); // сила по y, действующая на точку i со стороны точки j
        }
        forces[i].f_x = G * points[i].m * sum_x; // общая сила, действующая на материальную точку i в направлении x
        forces[i].f_y = G * points[i].m * sum_y; // общая сила, действующая на материальную точку i в направлении y
    }
}
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!


// обновление данных точек
__device__ void simulationStep(MatPoint points[], const Direction forces[], const int points_start_for_thread, const int points_end_for_thread) {
    for (int i = points_start_for_thread; i < points_end_for_thread; ++i) { // цикл по точкам, соответствующим потоку
        points[i].vx += forces[i].f_x / points[i].m * dt; // изменяем скорость по направлению x
        points[i].vy += forces[i].f_y / points[i].m * dt; // изменяем скорость по направлению y
        points[i].x += points[i].vx * dt; // изменяем положение по x
        points[i].y += points[i].vy * dt; // изменяем положение по y
    }
}


// вывод результатов в формате "время x_1 y_1 x_2 y_2 ..."
__device__ void print_results(const double t, const MatPoint points[], const int points_per_thread) {
    printf("%.3f ", t);
    for(int i=0; i < blockDim.x * points_per_thread; ++i) // проходим по всем точкам
        printf("%f %f ", points[i].x, points[i].y);
    printf("\n");
}


__global__ void Routine(MatPoint points[], Direction forces[], const int points_per_thread){ // функция, запускаемая на девайсе
    const int points_start_for_thread = threadIdx.x * points_per_thread; // с какой точки работает поток
    const int points_end_for_thread = (threadIdx.x + 1) * points_per_thread; // по какую точку
    while (t < t_end) {
        calcForce(points, forces, points_start_for_thread, points_end_for_thread); // считаем силы, действующие на все точки
        __syncthreads(); // синхронизируемся перед обновлением данных точек
        simulationStep(points, forces, points_start_for_thread, points_end_for_thread); // обновляем данные точек
        __syncthreads(); // синхронизируемся перед следующим шагом
        if (threadIdx.x == 0) { // поток 0 выводит данные и обновляет время
            print_results(t, points, points_per_thread); // выводим результат на шаге
            t += dt; // увеличиваем время
        }
        // __syncthreads();
    }
}


int main(int argc, char* argv[]) {
    if (argc == 2) // проверка наличия аргумента (сама программа + путь для считывания файла)
        input_path = argv[1];
    // считаем число строк в файле ==> столько будет потоков
    ifstream file1(input_path);
    int thread_count = 0;
    string line;
    while (getline(file1, line))
        ++thread_count;
    file1.close();

    hipMallocManaged(&points, thread_count * sizeof(MatPoint)); // выделение памяти под точки
    hipMallocManaged(&forces, thread_count * sizeof(Direction)); // выделение памяти по действующие силы


//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    if (thread_count > max_threads){ // проверка на лимит потоков
        points_per_thread = thread_count / max_threads; // изменяем число точек на поток
        thread_count = max_threads; // изменяем число потоков
    }
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!


    // заполнение массивов точек и действующих сил
    ifstream file2(input_path);
    double x, y, vx, vy, m;
    int iter = 0;
    while (!file2.eof()) { // один поток - одна точка
        file2 >> x >> y >> vx >> vy >> m; // берём данные из файла
        points[iter] = {x, y, vx, vy, m}; // заполняем данные для точек
        forces[iter] = {0, 0}; // заполняем данные для точек
        iter++;
    }
    file2.close();

    auto start = high_resolution_clock::now();

//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    Routine<<<block_count, thread_count>>>(points, forces, points_per_thread); // вызов девайсной функции (передаём число блоков и число потоков в блоке)
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    hipDeviceSynchronize();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(stop - start);
    // cout << fixed << setprecision(12) << duration.count() * 1e-9 << endl;

    hipFree(points); // освобождение памяти
    hipFree(forces);

    return 0;
}