
#include <hip/hip_runtime.h>
#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>
#include <vector>
#include <utility>
#include <fstream>

const double G = 6.674e-11; // гравитационная постоянная
const double dt = 0.001; // шаг по времени
const double e = 0.001; // чтобы сила не ушла в бесконечность

class MatPoint { // класс для материальной точки
    public:
        double x; // координата по x
        double y; // координата по y
        double vx; // скорость по x
        double vy; // скорость по y
        double m; // масса
};

std::vector<std::pair<double, double>> calcForce(const std::vector<MatPoint>& points) { // функция для пересчёта сил, действующих на материальную точку (на вход - вектор точек)
    std::vector<std::pair<double, double>> forces(points.size()); // вектор результата подсчитанных сил для всех точек по направлению x или y
    for (unsigned i = 0; i < points.size(); i+=1) { // проходим по всем точкам
        double sum_x = 0; // сумма сил, действующих на точку в направлении x
        double sum_y = 0; // сумма сил, действующих на точку в направлении y
        for (unsigned j = 0; j < points.size(); ++j) { // проход по всем другим точкам
            if (i == j) { // не рассматриваем одинаковые точки
                continue;
            }
            //================начальный вариант
            // double dist = sqrt(pow((points[j].x - points[i].x), 2) + pow((points[j].y - points[i].y), 2)); // рассчёт дистанции между точками
            // sum_x += points[j].m * (points[j].x - points[i].x) / pow(dist, 3); // суммируем силы по направлению x
            // sum_y += points[j].m * (points[j].y - points[i].y) / pow(dist, 3); // суммируем силы по направлению y
            //----------------добавление e в знаменатель
            double dist = sqrt(pow((points[j].x - points[i].x), 2) + pow((points[j].y - points[i].y), 2));
            sum_x += points[j].m * (points[j].x - points[i].x) / (pow(dist, 3) + e);
            sum_y += points[j].m * (points[j].y - points[i].y) / (pow(dist, 3) + e);
            //================
        }
        forces[i].first = G * points[i].m * sum_x;
        forces[i].second = G * points[i].m * sum_y;
    }
    return forces;
}

// шаг прохода
void simulationStep(std::vector<MatPoint>& points, const std::vector<std::pair<double, double>>& forces) {
    for (unsigned i = 0; i < points.size(); i+=1) { // проход по всем материальным точкам
        points[i].vx += forces[i].first / points[i].m * dt; // изменяем скорость по направлению x
        points[i].vy += forces[i].second / points[i].m * dt; // изменяем скорость по направлению y
        points[i].x += points[i].vx * dt; // изменяем положение по x
        points[i].y += points[i].vy * dt; // изменяем положение по y
    }
}

void read_file(std::vector<MatPoint>& points) { // заполнение вектора точек, считывание формата "x y v_x v_y m"
    std::ifstream file("input/input.txt");
    double x, y, vx, vy, m;
    while (!file.eof()) {
        file >> x >> y >> vx >> vy >> m;
        points.push_back({x, y, vx, vy, m});
    }
}

// вывод результатов в формате "время x_1 y_1 x_2 y_2 ..."
//===============================
void print_results(std::ofstream& file, double t, const std::vector<MatPoint> points) {
    file << t << " ";
    for (const auto &point: points) {
        file << point.x << "," << point.y << ", "; 
    }
    file << "\n";
}
//-------------------------------
// void print_results(double t, const std::vector<MatPoint> points) {
//     printf("%f ", t);
//     for (const auto &point: points) {
//         printf("%f %f ", point.x, point.y);
//     }
// }
//===============================

int main() {
    std::vector<MatPoint> points; // создание объекта для хранения информации о точках
    read_file(points); // взятие данных из файла

    std::ofstream file("output/output.txt");
    file << "t\t";
    for (unsigned i = 0; i < points.size(); ++i) {
        file << "x " << i + 1 << " y " << i + 1 << "\t";
    }
    file << "\n";

    double t = 0; // начальное время
    while(t < 20) { // цикл по времени 
        auto forces = calcForce(points); // считаем силы, действующие на все точки
        simulationStep(points, forces); // делаем шаг
        print_results(file, t, points);
        // print_results(t, points); // выводим результат на шаге
        t += dt; // увеличиваем время
    }
    return 0;
}
