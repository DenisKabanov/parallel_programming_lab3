
#include <hip/hip_runtime.h>
#include <iostream>
#define _USE_MATH_DEFINES
#include <vector>
#include <utility>
#include <fstream>
#include <string>
#include <cmath>
#include <chrono>
#include <iomanip>

using namespace std::chrono;

std::string input_path = "input/input.txt"; // путь для считывания начальных данных
std::string output_path = "output/output.txt"; // путь для записи полученных результатов

const double G = 6.674e-11; // гравитационная постоянная
const double dt = 0.001; // шаг по времени
const double e = 0.01; // чтобы сила не ушла в бесконечность

class MatPoint { // класс для материальной точки
    public:
        double x; // координата по x
        double y; // координата по y
        double vx; // скорость по x
        double vy; // скорость по y
        double m; // масса
};

std::vector<std::pair<double, double>> calcForce(const std::vector<MatPoint>& points) { // функция для пересчёта сил, действующих на материальную точку (на вход - вектор точек)
    std::vector<std::pair<double, double>> forces(points.size()); // вектор результата подсчитанных сил для всех точек по направлению x или y
    for (unsigned i = 0; i < points.size(); i+=1) { // проходим по всем точкам
        double sum_x = 0; // сумма сил, действующих на точку в направлении x
        double sum_y = 0; // сумма сил, действующих на точку в направлении y
        for (unsigned j = 0; j < points.size(); ++j) { // проход по всем другим точкам
            if (i == j) { // не рассматриваем одинаковые точки
                continue;
            }
            //================начальный вариант
            // double dist = sqrt(pow((points[j].x - points[i].x), 2) + pow((points[j].y - points[i].y), 2)); // рассчёт дистанции между точками
            // sum_x += points[j].m * (points[j].x - points[i].x) / pow(dist, 3); // суммируем силы по направлению x
            // sum_y += points[j].m * (points[j].y - points[i].y) / pow(dist, 3); // суммируем силы по направлению y
            //----------------добавление e в знаменатель
            double dist = sqrt(pow((points[j].x - points[i].x), 2) + pow((points[j].y - points[i].y), 2));
            sum_x += points[j].m * (points[j].x - points[i].x) / (pow(dist, 3) + e);
            sum_y += points[j].m * (points[j].y - points[i].y) / (pow(dist, 3) + e);
            //================
        }
        forces[i].first = G * points[i].m * sum_x;
        forces[i].second = G * points[i].m * sum_y;
    }
    return forces;
}

// шаг прохода
void simulationStep(std::vector<MatPoint>& points, const std::vector<std::pair<double, double>>& forces) {
    for (unsigned i = 0; i < points.size(); i+=1) { // проход по всем материальным точкам
        points[i].vx += forces[i].first / points[i].m * dt; // изменяем скорость по направлению x
        points[i].vy += forces[i].second / points[i].m * dt; // изменяем скорость по направлению y
        points[i].x += points[i].vx * dt; // изменяем положение по x
        points[i].y += points[i].vy * dt; // изменяем положение по y
    }
}

void read_file(std::vector<MatPoint>& points, std::string input_path = "input/input.txt") { // заполнение вектора точек, считывание формата "x y v_x v_y m"
    std::ifstream file(input_path);
    double x, y, vx, vy, m;
    while (!file.eof()) {
        file >> x >> y >> vx >> vy >> m;
        points.push_back({x, y, vx, vy, m});
    }
}

// вывод результатов в формате "время x_1 y_1 x_2 y_2 ..."
//===============================
void print_results(std::ofstream& file, double t, const std::vector<MatPoint> points) {
    file << t << " ";
    for (const auto &point: points) {
        file << point.x << " " << point.y << " "; 
    }
    file << "\n";
}
//-------------------------------
// void print_results(double t, const std::vector<MatPoint> points) {
//     printf("%f ", t);
//     for (const auto &point: points) {
//         printf("%f %f ", point.x, point.y);
//     }
// }
//===============================

int main(int argc, char* argv[]) {
    if (argc == 3) { // проверка наличия аргумента (сама программа + путь для считывания файла + путь для записи)
        input_path = argv[1];
        output_path = argv[2];
    }
    std::vector<MatPoint> points; // создание объекта для хранения информации о точках
    read_file(points, input_path); // взятие данных из файла

    auto start = high_resolution_clock::now();

    std::ofstream file(output_path);
    double t = 0; // начальное время
    while(t < 20) { // цикл по времени 
        auto forces = calcForce(points); // считаем силы, действующие на все точки
        simulationStep(points, forces); // делаем шаг
        print_results(file, t, points); // выводим результат на шаге
        t += dt; // увеличиваем время
    }
    file.close();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(stop - start);
    // std::cout << std::fixed << std::setprecision(12) << duration.count() * 1e-9 << std::endl;

    return 0;
}
